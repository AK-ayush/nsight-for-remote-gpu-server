#include "hip/hip_runtime.h"
/*
 * file2.cu
 *
 *  Created on: 06-Aug-2018
 *      Author: Ayush Kumar
 */




#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

/* In general it's good to include also the header of the current .c,
   to avoid repeating the prototypes */
#include "file2.h"

int deviceInfo(void)
{
  hipDeviceProp_t prop;
  int count,i;
  hipGetDeviceCount(&count);
  for(i=0;i<count;i++){

    hipGetDeviceProperties(&prop, i);
    printf("Device #%d ---------------\n",i);
    std::cout<<"Name: "<<prop.name<<"\n";
    std::cout<<"multiProcessorCount :"<<prop.multiProcessorCount<<"\n";
    std::cout<<"max thread per block :"<<prop.maxThreadsPerBlock<<"\n";
  }
return count;
}
